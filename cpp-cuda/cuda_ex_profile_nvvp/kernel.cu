#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

__global__ void copyKernel(int* src, int* dst)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	dst[idx] = src[idx];
}

int main()
{
	int* a_dev;
	int* b_dev;
	int* a = new int[128 * 100];
	int* b = new int[128 * 100];
	hipSetDevice(0);

	hipMalloc(&a_dev, sizeof(int) * 128 * 100);
	hipMalloc(&b_dev, sizeof(int) * 128 * 100);

	hipMemcpy(a_dev, a, sizeof(int) * 1000, hipMemcpyHostToDevice);

	copyKernel << <100, 128 >> > (a_dev, b_dev);
	hipDeviceSynchronize();

	hipMemcpy(b, b_dev, sizeof(int) * 1000, hipMemcpyDeviceToHost);

}